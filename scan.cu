#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

void cudaScanThrust(int *host_data, int len)
{
    thrust::device_ptr<int> device_data = thrust::device_malloc<int>(len);

    hipMemcpy(device_data.get(), host_data, len * sizeof(int),
               hipMemcpyHostToDevice);

    thrust::inclusive_scan(device_data, device_data + len, device_data);

    hipDeviceSynchronize();

    hipMemcpy(host_data, device_data.get(), len * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(device_data);
}


